#include "hip/hip_runtime.h"
// Copyright 2023 The gVisor Authors.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <hip/hip_runtime.h>
#include <err.h>
#include <errno.h>
#include <stdlib.h>
#include <unistd.h>

#include <cstdint>
#include <iostream>
#include <random>

#include "cuda_test_util.h"  // NOLINT(build/include)

__global__ void addKernel(std::uint32_t* data) {
  size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  data[index] += static_cast<std::uint32_t>(index);
}

void TestMallocManagedRoundTrip(int device, unsigned int malloc_flags,
                                bool prefetch) {
  constexpr size_t kNumBlocks = 32;
  constexpr size_t kNumThreads = 64;
  constexpr size_t kNumElems = kNumBlocks * kNumThreads;

  std::uint32_t* data = nullptr;
  constexpr size_t kNumBytes = kNumElems * sizeof(*data);
  CHECK_CUDA(hipMallocManaged(&data, kNumBytes, malloc_flags));

  // Initialize all elements in the array with a random value on the host.
  std::random_device rd;
  const std::uint32_t init_val =
      std::uniform_int_distribution<std::uint32_t>()(rd);
  for (size_t i = 0; i < kNumElems; i++) {
    data[i] = init_val;
  }

  if (prefetch) {
    CHECK_CUDA(hipMemPrefetchAsync(data, kNumBytes, device));
  }

  // Mutate the array on the device.
  addKernel<<<kNumBlocks, kNumThreads>>>(data);
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());

  if (prefetch) {
    CHECK_CUDA(hipMemPrefetchAsync(data, kNumBytes, hipCpuDeviceId));
  }

  // Check that the array has the expected result.
  for (size_t i = 0; i < kNumElems; i++) {
    std::uint32_t want = init_val + static_cast<std::uint32_t>(i);
    if (data[i] != want) {
      std::cout << "data[" << i << "]: got " << data[i] << ", wanted " << want
                << " = " << init_val << " + " << i << std::endl;
      abort();
    }
  }

  CHECK_CUDA(hipFree(data));
}

void TestMallocManagedReadWrite(int device) {
  constexpr size_t kNumBlocks = 32;
  constexpr size_t kNumThreads = 64;
  constexpr size_t kNumElems = kNumBlocks * kNumThreads;

  std::uint32_t* data = nullptr;
  constexpr size_t kNumBytes = kNumElems * sizeof(*data);
  CHECK_CUDA(hipMallocManaged(&data, kNumBytes, hipMemAttachGlobal));

  // Initialize all elements in the array with a random value on the host.
  std::random_device rd;
  const std::uint32_t init_val =
      std::uniform_int_distribution<std::uint32_t>()(rd);
  for (size_t i = 0; i < kNumElems; i++) {
    data[i] = init_val;
  }

  // Write the array's contents to a temporary file.
  char filename[] = "/tmp/cudaMallocManagedTest.XXXXXX";
  int fd = mkstemp(filename);
  if (fd < 0) {
    err(1, "mkstemp");
  }
  size_t done = 0;
  while (done < kNumBytes) {
    ssize_t n = write(fd, reinterpret_cast<char*>(data) + done,
                      kNumBytes - done);
    if (n >= 0) {
      done += n;
    } else if (n < 0 && errno != EINTR) {
      err(1, "write");
    }
  }

  // Mutate the array on the device.
  addKernel<<<kNumBlocks, kNumThreads>>>(data);
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());

  // Check that the array has the expected result.
  for (size_t i = 0; i < kNumElems; i++) {
    std::uint32_t want = init_val + static_cast<std::uint32_t>(i);
    if (data[i] != want) {
      std::cout << "data[" << i << "]: got " << data[i] << ", wanted " << want
                << " = " << init_val << " + " << i << std::endl;
      abort();
    }
  }

  // Read the array's original contents back from the temporary file.
  if (lseek(fd, 0, SEEK_SET) < 0) {
    err(1, "lseek");
  }
  done = 0;
  while (done < kNumBytes) {
    ssize_t n = read(fd, reinterpret_cast<char*>(data) + done,
                     kNumBytes - done);
    if (n > 0) {
      done += n;
    } else if (n == 0) {
      errx(1, "read: unexpected EOF after %zu bytes", done);
    } else if (n < 0 && errno != EINTR) {
      err(1, "read");
    }
  }

  // Check that the array matches what we originally wrote.
  for (size_t i = 0; i < kNumElems; i++) {
    std::uint32_t want = init_val;
    if (data[i] != want) {
      std::cout << "data[" << i << "]: got " << data[i] << ", wanted " << want
                << " = " << init_val << " + " << i << std::endl;
      abort();
    }
  }

  // Mutate the array on the device again.
  addKernel<<<kNumBlocks, kNumThreads>>>(data);
  CHECK_CUDA(hipGetLastError());
  CHECK_CUDA(hipDeviceSynchronize());

  // Check that the array has the expected result again.
  for (size_t i = 0; i < kNumElems; i++) {
    std::uint32_t want = init_val + static_cast<std::uint32_t>(i);
    if (data[i] != want) {
      std::cout << "data[" << i << "]: got " << data[i] << ", wanted " << want
                << " = " << init_val << " + " << i << std::endl;
      abort();
    }
  }

  close(fd);
  CHECK_CUDA(hipFree(data));
}

int main() {
  int device;
  CHECK_CUDA(hipGetDevice(&device));

  std::cout << "Testing hipMallocManaged(flags=hipMemAttachGlobal)"
            << std::endl;
  TestMallocManagedRoundTrip(device, hipMemAttachGlobal, false);

  int cma = 0;
  CHECK_CUDA(
      hipDeviceGetAttribute(&cma, hipDeviceAttributeConcurrentManagedAccess, device));
  if (!cma) {
    std::cout << "hipDeviceAttributeConcurrentManagedAccess not available"
              << std::endl;
  } else {
    std::cout << "Testing hipMallocManaged(flags=hipMemAttachGlobal) "
                 "with prefetching"
              << std::endl;
    TestMallocManagedRoundTrip(device, hipMemAttachGlobal, true);
    std::cout << "Testing hipMallocManaged(flags=hipMemAttachHost)"
              << std::endl;
    TestMallocManagedRoundTrip(device, hipMemAttachHost, false);
    std::cout << "Testing hipMallocManaged(flags=hipMemAttachHost) "
                 "with prefetching"
              << std::endl;
    TestMallocManagedRoundTrip(device, hipMemAttachHost, true);
  }

  std::cout << "Testing read/write syscalls on hipMallocManaged memory"
            << std::endl;
  TestMallocManagedReadWrite(device);

  std::cout << "All tests passed" << std::endl;
  return 0;
}
