// Copyright 2024 The gVisor Authors.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

// This program lists the features of the CUDA device that are available.
// It is used as part of the list_features.sh script.
// Each line it outputs is a CUDA feature name, prefixed by either
// "PRESENT: " or "ABSENT: ".

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

#include "cuda_test_util.h"  // NOLINT(build/include)

void printFeature(const char* feature, bool have) {
  if (have) {
    printf("PRESENT: %s\n", feature);
  } else {
    printf("ABSENT: %s\n", feature);
  }
}

int main(int argc, char *argv[]) {
  int cuda_device;
  CHECK_CUDA(hipGetDevice(&cuda_device));
  hipDeviceProp_t properties;
  CHECK_CUDA(hipGetDeviceProperties(&properties, cuda_device));
  bool cdpCapable =
      (properties.major == 3 && properties.minor >= 5) || properties.major >= 4;
  printFeature("DYNAMIC_PARALLELISM", cdpCapable);
  printFeature(
      "PERSISTENT_L2_CACHING", properties.persistingL2CacheMaxSize > 0);
  // Tensor cores are a thing in Volta (SM8X)
  printFeature("TENSOR_CORES", properties.major >= 8);
  int isCompressionAvailable;
  CHECK_CUDA_RESULT(
      hipDeviceGetAttribute(&isCompressionAvailable,
                           CU_DEVICE_ATTRIBUTE_GENERIC_COMPRESSION_SUPPORTED,
                           cuda_device));
  printFeature("COMPRESSIBLE_MEMORY", isCompressionAvailable != 0);
  bool p2pAvailable = false;
  int gpuCount = -1;
  CHECK_CUDA(hipGetDeviceCount(&gpuCount));
  printf("// Number of GPUs: %d\n", gpuCount);
  if (gpuCount >= 2) {
    int canAccessAToB = -1;
    CHECK_CUDA(hipDeviceCanAccessPeer(&canAccessAToB, 0, 1));
    printf("// CUDA P2P: 0 -> 1: %d\n", canAccessAToB);
    int canAccessBToA = -1;
    CHECK_CUDA(hipDeviceCanAccessPeer(&canAccessBToA, 1, 0));
    printf("// CUDA P2P: 1 -> 0: %d\n", canAccessBToA);
    p2pAvailable = canAccessAToB > 0 && canAccessBToA > 0;
  }
  printFeature("P2P", p2pAvailable);
}
